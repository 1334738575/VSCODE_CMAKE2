
#include <hip/hip_runtime.h>
// #include "CUDACommon.h"

// namespace VSCODE_CMAKE
// {
//     __device__ void add2(const float2 &a, const float2 &b, float2 &c)
//     {
//         c.x = a.x + b.x;
//         c.y = a.y + b.y;
//     }

//     __global__ void add2sCU(float2 *as, float2 *bs, float2 *cs, unsigned int s)
//     {
//         unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
//         if (idx >= s)
//             return;
//         add2(as[idx], bs[idx], cs[idx]);
//     }
//     void testAdd2s(float2 *as, float2 *bs, float2 *cs, unsigned int s)
//     {
//         dim3 block(1024, 1);
//         dim3 grid((s + 1023) / 1024, 1);
//         add2sCU<<<grid, block>>>(as, bs, cs, s);
//     }

// } // namespace VSCODE_CMAKE
